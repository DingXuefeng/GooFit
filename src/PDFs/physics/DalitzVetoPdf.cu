#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/DalitzPlotHelpers.h>
#include <goofit/PDFs/physics/DalitzVetoPdf.h>
#include <goofit/PDFs/physics/Amp3BodySqDP.h>

namespace GooFit {

__device__ auto device_DalitzVeto(fptype *evt, ParameterContainer &pc) -> fptype {
    int idx1 = pc.getObservable(0);
    int idx2 = pc.getObservable(1);

    fptype motherM = pc.getParameter(0);
    fptype d1m     = pc.getParameter(1);
    fptype d2m     = pc.getParameter(2);
    fptype d3m     = pc.getParameter(3);

    fptype mprime = evt[idx1];
    fptype thprime = evt[idx2];

    fptype m12 = calc_m12(mprime,motherM,d1m,d2m,d3m);
    fptype m13 = calc_m13(m12,cos(thprime*M_PI), motherM,d1m,d2m,d3m);
    fptype s12 = m12*m12;
    fptype s13 = m13*m13;
    fptype s23 = c_motherMass * c_motherMass + c_daug1Mass * c_daug1Mass + c_daug2Mass * c_daug2Mass
                 + c_daug3Mass * c_daug3Mass - s12 - s13;

    fptype x = s13;
    fptype y = s23;
    fptype z = s12;

    fptype ret    = inDalitz2(x, y, motherM, d1m, d2m, d3m) ? 1.0 : 0.0;
    int numVetos  = pc.getConstant(0);
    int numParams = pc.getNumParameters();
    int numCons   = pc.getNumConstants();
    int numObs    = pc.getNumObservables();
    int numNorms  = pc.getNumNormalizations();

    for(int i = 0; i < numVetos; ++i) {
        unsigned int varIndex = pc.getConstant(1 + i);
        fptype minimum        = pc.getParameter(4 + i * 2);
        fptype maximum        = pc.getParameter(4 + i * 2 + 1);
        fptype currDalitzVar  = (PAIR_13 == varIndex ? x : PAIR_23 == varIndex ? y : z);

        ret *= ((currDalitzVar < maximum) && (currDalitzVar > minimum)) ? 0.0 : 1.0;
    }

    // TODO: Prefer this function, not incrementIndex();
    // pc.incrementIndex(1, numParams, numVetos, numObs, numNorms);
    pc.incrementIndex();
    return ret;
}

__device__ device_function_ptr ptr_to_DalitzVeto = device_DalitzVeto;

__host__ DalitzVetoPdf::DalitzVetoPdf(std::string n,
                                      Observable _x,
                                      Observable _y,
                                      Variable motherM,
                                      Variable d1m,
                                      Variable d2m,
                                      Variable d3m,
                                      std::vector<VetoInfo> vetos)
    : GooPdf("DalitzVetoPdf", n, _x, _y, motherM, d1m, d2m, d3m) {
    registerConstant(vetos.size());

    for(auto &veto : vetos) {
        registerParameter(veto.minimum);
        registerParameter(veto.maximum);
        registerConstant(veto.cyclic_index);
    }

    registerFunction("ptr_to_DalitzVeto", ptr_to_DalitzVeto);

    initialize();
}
} // namespace GooFit
