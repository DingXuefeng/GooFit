#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/combine/MappedPdf.h>

namespace GooFit {

__device__ fptype device_Mapped(fptype *evt, ParameterContainer &pc) {
    // Structure : nP mapFunctionIndex mapParamIndex functionIndex1 parameterIndex1 functionIndex2 parameterIndex2 ...

    // Find mapping between event variables and function to evaluate
    unsigned int numTargets = pc.getConstant(0);
    auto numConstants = pc.getNumConstants();

    auto pc_mapped = pc;

    // Mapping PDF happens directly after, so just increment.
    // pc.incrementIndex (1, 0, 1, 0, 1);
    pc.incrementIndex();

    // This is an index into the MappedPdf's list of functions
    // int targetFunction = (int) floor(0.5 +
    // (*(reinterpret_cast<device_function_ptr>(d_function_table[mapFunction])))(evt, p, paramIndices +
    // indices[2]));
    auto targetFunction = static_cast<int>(floor(0.5 + callFunction(evt, pc)));
    int indicestoskip = 0;
    for(int i = 0; i < targetFunction; i++ ) {
        indicestoskip += (int) pc_mapped.getConstant(1 + i);
    }
    //printf("indices to skip %d %d \n", targetFunction, indicestoskip );
   
    auto cur_funcIdx = pc.funcIdx;
    auto target_funcIdx = cur_funcIdx + indicestoskip;
    while(pc.funcIdx < target_funcIdx) pc.incrementIndex();
    //printf("index of fucntion %d \n", pc.funcIdx); 


    // targetFunction *= 2; // Because there are two pieces of information about each function
    // targetFunction += 3; // Because first function information begins at index 3

    // Unsure these carry over...
    // targetFunction *= 2;
    // targetFunction += 3;

    // numTargets; increment past our set of mapping functions and our target functions after it is handled.
    unsigned int funcIdx = pc.funcIdx;

    /*
    if(targetFunction == 0) pc.funcIdx = 2;
    if(targetFunction == 1) pc.funcIdx = 20;
    // brute force to make it work
    if(targetFunction == 1) {
        pc.parameterIdx += 154 - 2;
        pc.constantIdx += 63 - 6;
        pc.observableIdx += 38 - 8;
        pc.normalIdx += 40 - 4;
    }
    */


    // fptype ret = (*(reinterpret_cast<device_function_ptr>(d_function_table[indices[targetFunction]])))(evt, p,
    // paramIndices + indices[targetFunction + 1]);
    fptype norm = pc.getNormalization(0);
    fptype ret  = callFunction(evt, pc);
    //std::cout << "mapped return: " << norm << " " << ret << std::endl;
    ret *= norm;

    // increment our functions here...
    /*
    if (pc.funcIdx==20) pc.funcIdx=38;
    if (pc.funcIdx==19) pc.funcIdx=38;
    //DOES THIS HELP?
    if(targetFunction == 0) {
        pc.parameterIdx = 306;
        pc.constantIdx = 120;
        pc.observableIdx = 68;
        pc.normalIdx = 76;
    }
    */
    int finalIndex = cur_funcIdx;
    //now need to increase index until end
    for(int i = 1; i < numConstants; i++ ) {
        finalIndex += (int) pc_mapped.getConstant(i);
    }

    while(pc.funcIdx < finalIndex) pc.incrementIndex();
    //printf("final index %d\n", pc.funcIdx);



    // if (gpuDebug & 1)
    // if ((gpuDebug & 1) && (0 == BLOCKIDX) && (0 == THREADIDX))
    // printf("[%i, %i] Mapped: %i (%f %f %f %f) %f\n", BLOCKIDX, THREADIDX, targetFunction, evt[0], evt[1], evt[2],
    // evt[3], ret);
    return ret;
}

__device__ device_function_ptr ptr_to_Mapped = device_Mapped;

__host__ int countComponents(PdfBase *func) {
    auto subcomponents = func->getComponents();
    int n_components = 0;
    if(subcomponents.size() > 0) {
      
      for(auto subcomponent : subcomponents) {
         n_components++;
         //printf("%s\n", subcomponent->getName().c_str());
        n_components += countComponents(subcomponent);
      }
    }
    
    return n_components;
}

__host__ MappedPdf::MappedPdf(std::string n, GooPdf *m, std::vector<GooPdf *> &t)
    : CombinePdf("MappedPdf", n) {
    components.push_back(m);

    std::vector<int> nComponents;

    for(GooPdf *f : t) {
        components.push_back(f);
        // pindices.push_back(f->getFunctionIndex());
        // pindices.push_back(f->getParameterIndex());
        // functionIndicesUsed.insert(f->getFunctionIndex());
        auto functionIndex = f->getFunctionIndex();

        //printf("function index %d\n", functionIndex);
        //for(auto subcomp : f->getComponents()) printf("subcomp\n");
        //count number of subfunctions
        int n_components = countComponents(f);
        //also count total function
        n_components++;
        nComponents.push_back(n_components);
        //printf("nubmer comp %d\n", n_components);
    }

    // if(functionIndicesUsed.size() > 1) {
    //    std::cout << "Warning: More than one function type given to MappedPdf " << getName()
    //              << " constructor. This may slow execution by causing sequential evaluations.\n";
    //}

    // This makes sure we have the appropriate amount of obs in our structure
    observablesList = getObservables();

    // add a constant value for the number of 't' functions, skipping 'm'.
    registerConstant(components.size() - 1);
    for(auto nComponent : nComponents) registerConstant(nComponent);

    registerFunction("ptr_to_Mapped", ptr_to_Mapped);

    initialize();
}

__host__ fptype MappedPdf::normalize() {
    // std::cout << "Normalizing MappedPdf " << getName() << std::endl;
    fptype ret = 0;

    for(unsigned int i = 1; i < components.size(); ++i) { // No need to normalize mapping function.
        fptype curr = components[i]->normalize();
        ret += curr;
    }

    host_normalizations[normalIdx + 1] = 1.0;
    cachedNormalization                = 1.0;

    return ret;
}
} // namespace GooFit
