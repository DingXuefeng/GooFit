#include "hip/hip_runtime.h"
#include <fstream>
#include <iomanip>
#include <iostream>
#include <string>
#include <vector>

#include "ChristophModel.h"

// GooFit stuff
#include "goofit/FitManager.h"
#include "goofit/PDFs/basic/PolynomialPdf.h"
#include "goofit/PDFs/combine/AddPdf.h"
#include "goofit/PDFs/physics/Tddp4Pdf.h"
#include "goofit/PDFs/physics/TruthResolution_Aux.h"
#include "goofit/UnbinnedDataSet.h"

namespace GooFit {
const fptype ChristophModel::D0_MASS = 1.8645;
const fptype ChristophModel::PI_MASS = 0.13957018;
const fptype ChristophModel::K_MASS = 0.493677;
const fptype ChristophModel::D0_MESON_RADUIS = 5.0;
const fptype ChristophModel::D0_TAU = 0.4101;
const fptype ChristophModel::SQ_WS_TO_RS_RATE = 1.0 / sqrt(300.0);

ChristophModel::ChristophModel(const fptype xMixingValue,
                               const fptype yMixingValue,
                               const unsigned int modelMCEventsNorm,
                               bool special_integral)
    : _dk3piTau("tau", ChristophModel::D0_TAU),
      _dk3piXMixing("xmixing", xMixingValue),
      _dk3piYMixing("ymixing", yMixingValue),
      _dk3piSqWStoRSrate("SqWStoRSrate", ChristophModel::SQ_WS_TO_RS_RATE),
      _dk3piDecayInfo(_dk3piTau, _dk3piXMixing, _dk3piYMixing,
                      _dk3piSqWStoRSrate),
      _eff("constantEff", _modelVars, _coefficients, _offsets, 0) {
  _dk3piDecayInfo.meson_radius = ChristophModel::D0_MESON_RADUIS;
  _dk3piDecayInfo.particle_masses = {
      ChristophModel::D0_MASS, ChristophModel::PI_MASS, ChristophModel::PI_MASS,
      ChristophModel::K_MASS, ChristophModel::PI_MASS};
  _dk3piDecayInfo.amplitudes_B.insert(std::end(_dk3piDecayInfo.amplitudes_B),
                                      std::begin(_rs_amplitudes),
                                      std::end(_rs_amplitudes));
  _dk3piDecayInfo.amplitudes.insert(std::end(_dk3piDecayInfo.amplitudes),
                                    std::begin(_ws_amplitudes),
                                    std::end(_ws_amplitudes));

  _dp = new TDDP4("test_TD", _modelVars, _dk3piDecayInfo, &_dat, &_eff, 0,
                  modelMCEventsNorm,special_integral);
}

void ChristophModel::setXMixingRangeForFit(const fptype error,
                                           const fptype lowerLimit,
                                           const fptype upperLimit) {
  // constructing a Variable without giving a range, as is done when
  // initializing xmixing in the constructor,
  // sets fixed = true by default but we want to float xmixing if fitting
  _dk3piDecayInfo._xmixing.setFixed(false);
  _dk3piDecayInfo._xmixing.setError(error);
  _dk3piDecayInfo._xmixing.setLowerLimit(lowerLimit);
  _dk3piDecayInfo._xmixing.setUpperLimit(upperLimit);
}

void ChristophModel::setYMixingRangeForFit(const fptype error,
                                           const fptype lowerLimit,
                                           const fptype upperLimit) {
  // constructing a Variable without giving a range, as is done when
  // initializing ymixing in the constructor,
  // sets fixed = true by default but we want to float ymixing if fitting
  _dk3piDecayInfo._ymixing.setFixed(false);
  _dk3piDecayInfo._ymixing.setError(error);
  _dk3piDecayInfo._ymixing.setLowerLimit(lowerLimit);
  _dk3piDecayInfo._ymixing.setUpperLimit(upperLimit);
}

void ChristophModel::setModelMaxWeight(const fptype wmax) {
  _dp->setMaxWeight(wmax);
}

void ChristophModel::setGenerationOffset(const unsigned int generationOffset) {
  _dp->setGenerationOffset(generationOffset);
}

std::tuple<mcbooster::ParticlesSet_h, mcbooster::VariableSet_h,
           mcbooster::RealVector_h, mcbooster::BoolVector_h>
ChristophModel::generateSig(const int batchSize, const int seed) {
  return _dp->GenerateSig(batchSize, seed);
}

void ChristophModel::addEventToCurrentDataToFit(double m12, double m34,
                                                double cos12, double cos34,
                                                double phi, double dt,
                                                double sigmaT, int eventNum, double eff) {
  _model_m12.setValue(m12);
  _model_m34.setValue(m34);
  _model_cos12.setValue(cos12);
  _model_cos34.setValue(cos34);
  _model_phi.setValue(phi);
  _model_eventNumber.setValue(eventNum);
  _model_dtime.setValue(dt);
  _model_sigmat.setValue(sigmaT);
  _model_eff.setValue(eff);
  _currentDataToFit.addEvent();

  // std::cout<<"Added event with:"<<std::endl;
  // std::cout<<"    m12: "<<m12<<std::endl;
  // std::cout<<"    m34: "<<m34<<std::endl;
  // std::cout<<"  cos12: "<<cos12<<std::endl;
  // std::cout<<"  cos34: "<<cos34<<std::endl;
  // std::cout<<"    phi: "<<phi<<std::endl;
  // std::cout<<"     dt: "<<dt<<std::endl;
  // std::cout<<" sigmaT: "<<sigmaT<<std::endl;
  // std::cout<<"event #: "<<eventNum<<std::endl<<std::endl;

  // FIXME do we need to reset the Variable values after we are done adding the
  // event?
  // it seems the DataSet has to be backed by the same Variables as the model
}

void ChristophModel::fitCurrentData(unsigned int sampleNum,
                                    const std::string &outFile) {
  // build pdf
  std::cout << "Building PDF..." << std::endl;
  Variable constant("constant1", 1.0);
  Variable constant2("constant2", 1.0);
  std::vector<Variable> backgrVars = {constant};
  PolynomialPdf backgr("backgr", _model_m12, backgrVars);
  AddPdf signal("signal", constant2, _dp, &backgr);

  // set current data
  std::cout << "Setting current data..." << std::endl;
  signal.setData(&_currentDataToFit);
  _dp->setDataSize(_currentDataToFit.getNumEvents(), 9);

  // do fitting
  std::cout << "Fitting data (" << _currentDataToFit.getNumEvents()
            << " events)..." << std::endl;
  FitManager datapdf(&signal);
  // datapdf.setMaxCalls(10000);
  auto fitResults = datapdf.fit();

  // get results
  std::cout << "Retrieving fit results..." << std::endl;
  int convergedStatus =
      fitResults.IsValid() ? 1 : 0; // convert status to int to make it easier
                                    // to read/write fit log; 1 = converged, 0 =
                                    // did not converge
  // covStatus: 0 = not calculated, 1 = approx but not accurate, 2 = full matrix
  // but forced pos def, 3 = full accurate cov matrix according to minuit2 doc
  int covStatus = fitResults.UserState().CovarianceStatus();

  // write results to file
  // format is one line with dataSampleNum fittedXMixVal fittedXMixError
  // fittedYMixVal fittedYMixError convergedStatus covStatus
  std::cout << "Appending fit results to file " << outFile << "..."
            << std::endl;
  std::ofstream out;
  out.open(outFile.c_str(), std::ios::app);
  out.precision(10);
  out << sampleNum << " " << _dk3piDecayInfo._xmixing.getValue() << " "
      << _dk3piDecayInfo._xmixing.getError() << " "
      << _dk3piDecayInfo._ymixing.getValue() << " "
      << _dk3piDecayInfo._ymixing.getError() << " " << convergedStatus << " "
      << covStatus << std::endl;
  out.close();
}

ChristophModel::~ChristophModel() { delete _dp; }
} // end namespace GooFit
